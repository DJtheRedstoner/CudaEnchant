
#include <hip/hip_runtime.h>
//
// Created by DJtheRedstoner on 5/4/2021.
//

class SimpleRandom {
private:
    const static long long multiplier = 0x5DEECE66D;
    const static long long mask = (1LL << 48) - 1;
    long long seed = 0LL;
public:
    __device__
    void setSeed(long long newSeed) {
        seed = (newSeed ^ multiplier) & mask;
    }
    __device__
    int nextInt(int bound) {
        int r = next();
        int m = bound - 1;
        if ((bound & m) == 0)
            r = (int)((bound * (long long)r) >> 31);
        else {
            int u = r;
            while (u - (r = u % bound) + m < 0) u = next();
        }
        return r;
    }
private:
    __device__
    int next() {
        seed = (seed * multiplier + 0xBLL) & mask;
        return (int)(seed >> 17);
    }
};
