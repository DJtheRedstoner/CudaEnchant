#include "hip/hip_runtime.h"
//
// Created by DJtheRedstoner on 5/5/2021.
//

#include <iostream>
#include "SimpleRandom.cu"

__device__
inline int getGenericEnchantability(SimpleRandom& random, int bookshelves) {
    int first = random.nextInt(8);
    int second = random.nextInt(bookshelves + 1);
    return first + 1 + (bookshelves >> 1) + second;
}

__device__
inline int getLevelsSlot1(SimpleRandom& random, int bookshelves) {
    int enchantability = getGenericEnchantability(random, bookshelves) / 3;
    return enchantability < 1 ? 1 : enchantability;
}

__device__
inline int getLevelsSlot2(SimpleRandom& random, int bookshelves) {
    return getGenericEnchantability(random, bookshelves) * 2 / 3 + 1;
}

__device__
inline int getLevelsSlot3(SimpleRandom& random, int bookshelves) {
    int enchantability = getGenericEnchantability(random, bookshelves);
    int twiceBookshelves = bookshelves * 2;
    return enchantability < twiceBookshelves ? twiceBookshelves : enchantability;
}

__device__
inline bool checkSlots(int bookshelves, SimpleRandom r, const int* data) {
    int slot1 = data[bookshelves * 3];
    int slot2 = data[bookshelves * 3 + 1];
    int slot3 = data[bookshelves * 3 + 2];

    if (slot1 == 0) return true;

    if (getLevelsSlot1(r, bookshelves) == slot1) {
        if (getLevelsSlot2(r, bookshelves) == slot2) {
            if (getLevelsSlot3(r, bookshelves) == slot3) {
                return true;
            }
        }
    }
    return false;
}

__global__
void fullCrack(const int* data, long long* p_seed, int* counts) {

    SimpleRandom r;

    unsigned int first = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    int count;
    for (long long seed = first; seed < (1LL << 32) - 1; seed += stride) {
        for (int i = 15; i >= 0; i--) {
            r.setSeed((int) seed);
            if (!checkSlots(i, r, data)) goto fail;
        }
        //printf("%lli\n", seed);
        count++;
        *p_seed = seed;
        fail:
        ;
    }

    counts[first] = count;
}

bool initialized = false;
int* data;

void resetCracker() {
    if (initialized) {
        hipFree(data);
    }
    hipMallocManaged(&data, 16*3*sizeof(int));

    for (int i = 0; i < 16 * 3; i++) {
        data[i] = 0;
    }

    initialized = true;
}

void addInfo(int bookshelves, int slot1, int slot2, int slot3) {
    data[bookshelves * 3] = slot1;
    data[bookshelves * 3 + 1] = slot2;
    data[bookshelves * 3 + 2] = slot3;
}

int main() {

    int blockSize = 1024;
    int blockCount = 32;
    int threadCount = blockCount * blockSize;


    /***
     * 15 07 13 30 100662581
     * 14 07 14 28 1275608
     * 13 05 15 26 31570
     * 12 04 09 24 734
     * 11 04 11 22 55
     * 10 06 06 20 F767F75E
     */
    resetCracker();
    addInfo(15,  7, 13, 30);
    addInfo(14,  7, 14, 28);
    addInfo(13,  5, 15, 26);
    addInfo(12,  4,  9, 24);
    addInfo(11,  4, 11, 22);
    addInfo(10,  6,  6, 20);

    long long* seed;
    hipMallocManaged(&seed, sizeof(long long));

    int* count;
    hipMallocManaged(&count, threadCount*sizeof(int));

    fullCrack<<<blockCount, blockSize>>>(data, seed, count);
    hipDeviceSynchronize();

    int total = 0;
    for (int i = 0; i < threadCount; i++) {
        total += count[i];
    }

    std::cout << total << std::endl;
    printf("%llX", *seed);

    hipFree(data);
    hipFree(seed);
    hipFree(count);
}